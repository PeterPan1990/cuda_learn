//MatAdd.cu
// author: Pan Yang
// date  : 2015-7-4


#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h>
#include <time.h>
    
#define M 512  // height of A
#define N 512  // width of A ( == height of B)
#define P 512  // width of B

#define BLOCK_SIZE 32

typedef struct {
    int height;
    int width;
    float *elements;
}Matrix;
    
// cpu code definition 
void MatMulOnHost(const Matrix A, const Matrix B, Matrix C)
{
    int i, j, k;
    for (i = 0; i < A.height; ++i)
    {
        for (j = 0; j < B.width; ++j)
        {
            float sum_ij = 0;
            
            for (k = 0; k < A.width; ++k)
                sum_ij += A.elements[i * A.width + k] * B.elements[k * B.width + j];
            C.elements[i * C.width + j] = sum_ij;
        }
    }
}
  
    
// Kernel definition 
__global__ void MatMul(const Matrix A, const Matrix B, Matrix C)
{
    // Each thread computes one element of C by accumulating results into Cvalue
    float C_ij = 0; 
    int i = blockIdx.y * blockDim.y + threadIdx.y; 
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    int k;
    for (k = 0; k < A.width; ++k)
    {
        C_ij += A.elements[i * A.width + k] * B.elements[k * B.width + j];
    }
    
    C.elements[i * C.width + j] = C_ij;
}

int main() 
{ 
    // time for the whole process
    printf("\n");
    printf("[Matrix Multiply Using CUDA] - Starting...\n");
    int devID = 0;
    hipDeviceProp_t deviceProp;
    hipError_t error;
    
    error = hipGetDeviceProperties(&deviceProp, devID);
    if (error != hipSuccess)
    {
        printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
    }
    else
    {
        printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major,      deviceProp.minor);
    }
    
    int i;
    
    // load A, B, C on host
    Matrix A;
    A.height = M;
    A.width = N;
    size_t size = A.height * A.width * sizeof(float);
    A.elements = (float *)malloc(size);
    
    Matrix B;
    B.height = N;
    B.width = P;
    size = B.height * B.width * sizeof(float);
    B.elements = (float *)malloc(size);
    
    // initialize A
    srand(rand());
    for (i = 0; i < A.height * A.width; ++i)
    {
        A.elements[i] = rand() / (float)RAND_MAX;
        //A.elements[i] = 1.0;
    }
    // initialize B
    srand(rand());
    for (i = 0; i < B.height * B.width; ++i)
    {
        B.elements[i] = rand() / (float)RAND_MAX;
        //B.elements[i] = 0.01;
    }
    
    Matrix C;
    C.height = M;
    C.width = P;
    size = C.height * C.width * sizeof(float);
    C.elements = (float *)malloc(size); // default all zeros
    
    Matrix ref_C; // reference C for result check
    ref_C.height = M;
    ref_C.width = P;
    size = ref_C.height * ref_C.width * sizeof(float);
    ref_C.elements = (float *)malloc(size); // default all zeros
    
    // load a, b, c on device
    Matrix d_A;
    d_A.height = M;
    d_A.width = N;
    size = d_A.height * d_A.width * sizeof(float);
    hipMalloc(&d_A.elements, size); // alloc memory on device
    hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice); // copy data from host to device
    
    Matrix d_B;
    d_B.height = N;
    d_B.width = P;
    size = d_B.height * d_B.width * sizeof(float);
    hipMalloc(&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
    
    Matrix d_C;
    d_C.height = M;
    d_C.width = P;
    size = d_C.height * d_C.width * sizeof(float);
    hipMalloc(&d_C.elements, size); // default are not zeros, maybe random number
    
    // Kernel invocation with m*16*16 threads
    dim3 BlockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 GridDim((P + BlockDim.x - 1) / BlockDim.x, (M + BlockDim.x - 1) / BlockDim.y);
    
    hipEvent_t start_cu, stop_cu;
    float time_gpu = 0.0f;
    hipEventCreate(&start_cu);
    hipEventCreate(&stop_cu);
    hipEventRecord( start_cu, 0);
    
    int nIter = 300;
    
    for (i = 0; i < nIter; ++i)
    {
        MatMul<<<GridDim, BlockDim>>>(d_A, d_B, d_C);
    }
    hipEventRecord( stop_cu, 0);
    hipEventSynchronize( stop_cu );
    hipEventElapsedTime( &time_gpu, start_cu, stop_cu );
    hipEventDestroy( start_cu );
    hipEventDestroy( stop_cu );
    
    float msecPerMatrixMul = time_gpu / nIter;
    double flopsPerMatrixMul = 2.0 * (double)A.width * (double)A.height * (double)B.width;
    double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
    printf("Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops, WorkgroupSize= %u threads/block\n",
            gigaFlops,
            msecPerMatrixMul,
            flopsPerMatrixMul,
            BlockDim.x * BlockDim.y);
    
    // copy results form device memory to host memory
    hipMemcpy( C.elements, d_C.elements, size, hipMemcpyDeviceToHost );
  
    MatMulOnHost(A, B, ref_C);

    // test relative error by the formula
    //     |<x, y>_cpu - <x,y>_gpu| / <|x|, |y|>  < eps
    printf("Checking computed result for correctness ... \n");
    bool correct = true;
    
    double eps = 1.e-6 ; // machine zero

    for (int i = 0; i < (int)(M * P); i++)
    {
        double abs_err = fabs(ref_C.elements[i] - C.elements[i]);
        double dot_length = A.width;
        double abs_val = fabs(ref_C.elements[i]);
        double rel_err = abs_err / abs_val / dot_length ;

        if (rel_err > eps)
        {
            printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n", i, ref_C.elements[i], C.elements[i], eps);
            correct = false;
        }
    }
    
    printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");
  
    // free space
    free(C.elements); 
    free(B.elements); 
    free(A.elements);
    
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
    
    return 0;
}
